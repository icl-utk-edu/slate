#include "hip/hip_runtime.h"
// Copyright (c) 2017-2022, University of Tennessee. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
// This program is free software: you can redistribute it and/or modify it under
// the terms of the BSD 3-Clause license. See the accompanying LICENSE file.

#include "slate/Exception.hh"
#include "slate/internal/device.hh"

#include "device_util.cuh"

#include <cstdio>

namespace slate {
namespace device {

//------------------------------------------------------------------------------
/// Kernel implementing element-wise matrix reduction.
/// Each thread block deals with one tile.
/// Each thread deals with one row.
/// Launched by gereduce().
///
/// @param[in] m
///     Number of rows of each tile. m >= 1.
///
/// @param[in] n
///     Number of columns of each tile. n >= 1.
///
/// @param[in] mt
///     Number of tiles of A matrix. mt > 0.
///
/// @param[in] alpha
///     The scalar alpha.
///
/// @param[in] Aarray
///     Array of tiles of dimension mt-by-gridDim.x,
///     where each Aarray[k] is an m-by-n matrix stored in an lda-by-n array.
///
/// @param[in] lda
///     Leading dimension of each tile in Aarray. lda >= m.
///
/// @param[in] beta
///     The scalar alpha.
///
/// @param[in] Barray
///     Array of tiles of dimension 1-by-gridDim.x,
///     where each Barray[k] is an m-by-n matrix stored in an ldb-by-n array.
///
/// @param[in] ldb
///     Leading dimension of each tile in Aarray. lda >= m.
///
template <typename scalar_t>
__global__ void gereduce_kernel(
    int64_t m, int64_t n, int64_t mt,
    scalar_t alpha, scalar_t** Aarray, int64_t lda,
    scalar_t beta,  scalar_t** Barray, int64_t ldb)
{
    int64_t row = threadIdx.x;
    int64_t col = blockIdx.y;
    int64_t ib  = blockDim.x;
    int64_t jb  = gridDim.y;
    int64_t nt  = gridDim.x;
    scalar_t sum;

    // The first row of tiles.
    // All other rows will be sumed up to this row.
    scalar_t* B = Barray[ blockIdx.x ];


    // i, j loops sub-tile the tiles,
    for (int64_t i = row; i < m; i += ib) {
        for (int64_t j = col; j < n; j += jb) {
            // vector read into registers.
            // initial value of sum
            sum = beta * B[ j*ldb + i ];

            // Loop over block rows.
            for (int64_t ii = 0; ii < mt; ++ii) {
                // vector read and add
                scalar_t* Aii = Aarray[ ii*nt + blockIdx.x ];
                sum += alpha * Aii[ j*lda + i ];
            }

            // vector write
            B[ j*ldb + i ] = sum;
        }
    }
}

//------------------------------------------------------------------------------
/// Batched routine for element-wise matrix reduction.
/// Sets
/// \[
///     Barray[0, j] = (\sum_{i = 0:mt-1} \alpha Aarray[i, j]) +
///     \beta Barray[0, j]. j = 0:batch_count-1
/// \]
///
/// @param[in] m
///     Number of rows of each tile. m >= 0.
///
/// @param[in] n
///     Number of columns of each tile. n >= 0.
///
/// @param[in] mt
///     Number of tiles of A matrix. mt > 0.
///
/// @param[in] alpha
///     The scalar alpha.
///
/// @param[in] Aarray
///     Array in GPU memory of dimension batch_count, containing pointers to
///     tiles,
///     where each Aarray[i, j] is an m-by-n matrix stored in an lda-by-n array
///     in GPU memory.
///
/// @param[in] lda
///     Leading dimension of each tile in A. lda >= m.
///
/// @param[in] Barray
///     Brray in GPU memory of dimension mt-by-batch_count, containing pointers
///     to tiles,
///     where each Barray[0, j] is an m-by-n matrix stored in an lda-by-n array
///     in GPU memory.
///
/// @param[in] ldb
///     Leading dimension of each tile in B. ldb >= m.
///
/// @param[in] batch_count
///     Size of Aarray and Barray. batch_count >= 0.
///
/// @param[in] queue
///     BLAS++ queue to execute in.
///
template <typename scalar_t>
void gereduce(
    int64_t m, int64_t n, int64_t mt,
    scalar_t alpha, scalar_t** Aarray, int64_t lda,
    scalar_t beta,  scalar_t** Barray, int64_t ldb,
    int64_t batch_count, blas::Queue &queue)
{
    // quick return
    if (batch_count == 0)
        return;

    //int64_t nt = ceildiv( batch_count, mt );
    int64_t nt = batch_count;
    int64_t jb = 32;

    // Max threads/block=1024 for current CUDA compute capability (<= 7.5)
    hipSetDevice( queue.device() );

    int64_t nthreads = std::min( int64_t( 1024 ), m );
    dim3 threads( nthreads );
    dim3 blocks( nt, jb );

    gereduce_kernel<<< blocks, threads, 0, queue.stream() >>>(
        m, n, mt,
        alpha, Aarray, lda,
        beta,  Barray, ldb);

    hipError_t error = hipGetLastError();
    slate_assert(error == hipSuccess);

}

//------------------------------------------------------------------------------
// Explicit instantiations.
template
void gereduce(
    int64_t m, int64_t n, int64_t mt,
    float alpha, float** Aarray, int64_t lda,
    float beta,  float** Barray, int64_t ldb,
    int64_t batch_count, blas::Queue &queue);

template
void gereduce(
    int64_t m, int64_t n, int64_t mt,
    double alpha, double** Aarray, int64_t lda,
    double beta,  double** Barray, int64_t ldb,
    int64_t batch_count, blas::Queue &queue);

template
void gereduce(
    int64_t m, int64_t n, int64_t mt,
    hipFloatComplex alpha, hipFloatComplex** Aarray, int64_t lda,
    hipFloatComplex beta,  hipFloatComplex** Barray, int64_t ldb,
    int64_t batch_count, blas::Queue &queue);

template
void gereduce(
    int64_t m, int64_t n, int64_t mt,
    hipDoubleComplex alpha, hipDoubleComplex** Aarray, int64_t lda,
    hipDoubleComplex beta,  hipDoubleComplex** Barray, int64_t ldb,
    int64_t batch_count, blas::Queue &queue);

} // namespace device
} // namespace slate
